#include <stdio.h>
#include <stdbool.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define B 4
#define N 1024
#define UPPER_BIT 31
#define LOWER_BIT 0

__device__ void partition_by_bit(unsigned int *values, unsigned int bit);


/*******************************************************************************
   RADIX_SORT()

   For each bit position from the least significant to the most significant,
   partition the elements so that all elements with a 0 in that bit position
   precede those with a 1 in that position, using a stable sort.
   When all bits have been so processed, the array is sorted.
   Reminder -- a sort is stable if the sort preserves the relative order of 
               equal elements.

   Because this is a device function (executed by each thread concurrently),
   after each partitioning step, the threads must execute __syncthreads() so
   that the array is guaranteed to be ready for the next step.
*******************************************************************************/

__global__ void radix_sort(unsigned int *values)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ unsigned int vals[N];

	if (gid < B*N) vals[tid] = values[gid];
    __syncthreads();
    
    for(int bit = LOWER_BIT; bit <= UPPER_BIT; ++bit )
    {
        partition_by_bit(vals, bit);
        __syncthreads();
    }
	
	if (gid < B*N) values[gid] = vals[tid];
}

/*******************************************************************************

   PLUS_SCAN()

   plus_scan(a[]), where a[] is an array of integers, replaces a[] by the prefix
   sums of the elements of a. The prefix sum of an element in an array (or more 
   generally, any sequence) is the sum of all elements up to and including that
   element. The sum operation can be replaced by any binary associative operator,
   such as multiplication.

   A thread with ID i that calls plus_scan(a) gets as its return value the new 
   element in a[i]. All threads together collectively replace the elements of
   a[].
   

   Example:
                   A =  3  1  7  0  4  1  6  3 

   Successive iterations yield
       offset = 1  A =  3  4  8  7  4  5  7  9
       offset = 2  A =  3  4 11 11 12 12 11 14
       offset = 4  A =  3  4 11 11 15 16 22 25

   When it is finished it will have taken log N steps and used N log N adds.
   (This means that it is not work-efficient, since the sequential algorithm
    uses N adds.)

*******************************************************************************/
template<class T>
__device__ T plus_scan(T *x)
{
    unsigned int i = threadIdx.x; // id of thread executing this instance
    unsigned int n = blockDim.x;  // total number of threads in this block
    unsigned int offset;          // distance between elements to be added

    for( offset = 1; offset < n; offset *= 2) {
        T t;

        if ( i >= offset ) t = x[i-offset];
        __syncthreads();

        if ( i >= offset ) x[i] = t + x[i];      // i.e., x[i] = x[i] + x[i-1]
        __syncthreads();
    }
    return x[i];
}

/*******************************************************************************
    partition_by_bit()

    This function is executed by every thread. Given an array of non-negative
    integer values, and a bit position, b, this partitions the array such that
    for all values[i], i = 0,...,n-1, the value of bit b in each element 
    values[k]  for k < i is <= the value of bit b in values[i], and if bit b in
    values[j] == bit b in values[i], and j < i, then after the partition, the 
    two elements will be in the same relative order (i.e., it is a stable sort).

    Each thread is responsible for repositioning a single element of the array.
*******************************************************************************/

__device__ 
void partition_by_bit(unsigned int *values, unsigned int bit)
{
    unsigned int i = threadIdx.x;
    unsigned int size = blockDim.x;
    unsigned int x_i = values[i];          // value of integer at position i
    unsigned int p_i = (x_i >> bit) & 1;   // value of bit at position bit

    // Replace values array so that values[i] is the value of bit bit in
    // element i.
    values[i] = p_i;  

    // Wait for all threads to finish this.
    __syncthreads();

    // Now the values array consists of 0's and 1's, such that values[i] = 0
    // if the bit at position bit in element i was 0 and 1 otherwise.

    // Compute number of True bits (1-bits) up to and including values[i], 
    // transforming values[] so that values[i] contains the sum of the 1-bits
    // from values[0] .. values[i]
    unsigned int T_before = plus_scan(values);
/*
    plus_scan(values) returns the total number of 1-bits for all j such that
    j <= i. This is assigned to T_before, the number of 1-bits before i 
    (includes i itself)
*/

    // The plus_scan() function does not return here until all threads have
    // reached the __syncthreads() call in the last iteration of its loop
    // Therefore, when it does return, we know that the entire array has had
    // the prefix sums computed, and that values[size-1] is the sum of all
    // elements in the array, which happens to be the number of 1-bits in 
    // the current bit position.
    unsigned int T_total  = values[size-1];
    // T_total, after the scan, is the total number of 1-bits in the entire array.

    unsigned int F_total  = size - T_total;
/*    
    F_total is the total size of the array less the number of 1-bits and hence
    is the number of 0-bits.
*/
    __syncthreads();

/*
    The value x_i must now be put back into the values array in the correct
    position. The array has to satisfy the condition that all values with a 0 in
    the current bit position must precede all those with a 1 in that position
    and it must be stable, meaning that if x_j and x_k both had the same bit 
    value before, and j < k, then x_j must precede x_k after sorting.

    Therefore, if x_i had a 1 in the current bit position before, it must now
    be in the position such that all x_j that had a 0 precede it, and all x_j
    that had a 1 in that bit and for which j < i, must precede it. Therefore
    if x_i had a 1, it must go into the index T_before-1 + F_total, which is the
    sum of the 0-bits and 1-bits that preceded it before (subtracting 1 since
    T_before includes x_i itself).

    If x_i has a 0 in the current bit position, then it has to be "slid" down
    in the array before all x_j such that x_j has a 1 in the current bit, but 
    no farther than that. Since there are T_before such j, it has to go to
    position i - T_before.  (There are T_before such j because x_i had a zero,
    so in the prefix sum, it does not contribute to the sum.)
*/
    if ( p_i ) values[T_before-1 + F_total] = x_i;
    else values[i - T_before] = x_i;
/*
   The interesting thing is that no two values will be placed in the same 
   position. I.e., this is a permutation of the array.

   Proof: Suppose that x_i and x_j both end up in index k. There are three
   cases: 
     Case 1. x_i and x_j have a 1 in the current bit position 
     Since F_total is the same for all threads, this implies that T_before must
     be the same for threads i and j. But this is not possible because one must 
     precede the other and therefore the one that precedes it must have smaller
     T_before.

     Case 2.  x_i and x_j both have a 0 in the current bit position. 
     Since they both are in k, we have 
         k = i - T_bef_i = j - T_Bef_j  or
         i - j = T_bef_i - T_bef_j
     Assume i > j without loss of generality.  This implies that the number of
     1-bits from position j+1 to position i-1 (since both x_j and x_i have 
     0-bits) is i-j. But that is impossible since there are only i-j-2 positions
     from j+1 to i-1.

     Case 3. x_i and x_j have different bit values. 
     Assume without loss of generality that x_j has the 0-bit and x_i, the 1-bit.
     T_before_j is the number of 1 bits in positions strictly less than j, 
     because there is a 0 in position j. The total number of positions less than
     j is j, since the array is 0-based. Therefore:

     j-T_before_j is the number of 0-bits in positions strictly less than j. 
     This must be strictly less than F_total, since x_j has a 0 in position j, 
     so there is at least one more 0 besides those below position j. Hence:

     (1)    F_total > j - T_before_j

     Turning to i, T_before_i is at least 1, since x_i has a 1 in its bit. So, 
     T_before_i - 1 is at least 0, and 

     (2)    T_before_i - 1 + F_total >= F_total. 

     Therefore, combining (1) and (2)

     (3)   T_before_i - 1 + F_total >= F_total  
                                    >  j - T_before_j

     But if x_i and x_j map to the same position, then 

     (4)   j - T_before_j  = T_before_i - 1 + F_total 
                           > j - T_before_j

     which is a contradiction since a number cannot be greater than itself!

     Therefore it is impossible for x_i and x_j to be placed in the same index
     if i != j.
     
*/
}

bool check_sort(unsigned int * arr, unsigned int len)
{
	for (unsigned int i = 0; i < len-1; i++)
		if (arr[i] > arr[i+1]) return false;
	return true;
}

int main()
{
	srand(time(NULL));
	unsigned int a[B*N], b[B*N];
	
	// provide sorted subsequences
	for(unsigned int i = 0; i < B*N; i++){ 
		a[i] = rand() % N;
	}

	// original array
	#ifdef DEBUG
	printf("\n");
	for(unsigned int k = 0; k < B; k++) {
		printf("ORIGINAL ARRAY %d: \n", k);
		for(unsigned int i = 0; i < N; i++) 
			printf("%u ", a[k*N+i]);
		printf("\n");
	}
	#endif
	
	// allocate
	unsigned int *dc;
	hipMalloc((void**)&dc, sizeof(unsigned int)*B*N);
	// copy
	hipMemcpy(dc, a, sizeof(unsigned int)*B*N, hipMemcpyHostToDevice);
	
	// kernel launch and copy back
	radix_sort<<<B, N>>>(dc);
	hipMemcpy(b, dc, sizeof(unsigned int)*B*N, hipMemcpyDeviceToHost);
	
	// sorted array
	#ifdef DEBUG
	printf("\n");
	for(unsigned int k = 0; k < B; k++) {
		printf("SORTED ARRAY %d: \n", k);
		for(unsigned int i = 0; i < N; i++) 
			printf("%u ", b[k*N+i]);
		printf("\n");
	}
	#endif

	#ifdef CHECK
	printf("\n");
	for(unsigned int k = 0; k < B; k++) {
		if (check_sort(b+k*N, N))
			printf("CHECK PASS %u\n", k);
		else
			printf("CHECK FAIL %u\n", k);
	}
    #endif
	
    hipFree(dc);
	hipDeviceReset();

	return 0;
}